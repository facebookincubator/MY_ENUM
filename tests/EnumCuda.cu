#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
//
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <stdio.h>

#include "../EnumO.h"

MY_ENUM_O(Fruits, int, (apple, banana, pear));

__global__ void add(int n, float a, float* x) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    x[i] = a * x[i];
  }
}

int main(int argc, char** args) {
  int N = 1 << 20;
  float *x, *d_x;
  x = (float*)malloc(N * sizeof(float));

  hipMalloc(&d_x, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
  }

  hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

  add<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x);

  hipFree(d_x);
  free(x);
}
